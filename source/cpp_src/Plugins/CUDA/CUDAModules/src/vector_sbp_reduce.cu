#include "hip/hip_runtime.h"
#include "vector_sbp_reduce.h"


static const int blockSize = 1024;

__global__ void cuda_vector_sbp_reduce(const float* a, float* out, int n)
{
    int idx = threadIdx.x;
    float sum = 0.0;

    for (int i = idx; i < n; i += blockSize)
    {
        sum += a[i];
    }

    __shared__ float r[blockSize];

    r[idx] = sum;
    __syncthreads();

    for (int size = blockSize/2; size>0; size/=2)
    {
        if (idx<size)
        {
            r[idx] += r[idx+size];
        }
        __syncthreads();
    }
    if (idx == 0){ *out = r[0];};
}

void vector_sbp_reduce(float* a, float* b, int n)
{

    // Device input vectors
    float* d_a;
    float* d_out;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_out, sizeof(float));

    // Copy host input vector to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);


    // Number of threads in each thread block is blockSize = 1024;
    // Number of thread blocks in grid
    int gridSize;
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    cuda_vector_sbp_reduce<<<gridSize, blockSize>>>(d_a, d_out, n);

    // Copy array back to host
    hipMemcpy(b, d_out, sizeof(float), hipMemcpyDeviceToHost );

    // Release device memory
    hipFree(d_a);
    hipFree(d_out);
};
