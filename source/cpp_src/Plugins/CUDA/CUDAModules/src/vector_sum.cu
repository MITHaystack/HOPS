#include "hip/hip_runtime.h"
#include "vector_sum.h"


//cuda vector addition A+B=C
__global__ void cuda_vector_add(float* a, float* b, float* c, int length)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < length)
    {
        c[id] = a[id] + b[id];
    }
}


void vector_sum(float* a, float* b, float* c, int n)
{

    // Device input vectors
    float* d_a;
    float* d_b;
    //Device output vector
    float* d_c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy host vectors to device
    hipMemcpy( d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    cuda_vector_add<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost );

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
};
