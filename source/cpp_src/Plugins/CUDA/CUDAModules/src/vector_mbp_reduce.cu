#include "hip/hip_runtime.h"
#include "vector_mbp_reduce.h"


__global__ void cuda_vector_mbp_reduce(const float* a, float* out, int n)
{
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;

    float sum = 0.0;

    for (int i = gthIdx; i < n; i += gridSize)
    {
        sum += a[i];
    }

    __shared__ float shArr[blockSize];
    shArr[thIdx] = sum;

    __syncthreads();

    for (int size = blockSize/2; size>0; size/=2)
    {
        if(thIdx<size)
        {
            shArr[thIdx] += shArr[thIdx+size];
        }
        __syncthreads();
    }

    if (thIdx == 0)
    {
        out[blockIdx.x] = shArr[0];
    }
}

void vector_mbp_reduce(float* a, float* b, int n)
{
    // Device input vectors
    float* d_a;
    float* d_out;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_out, blockSize*sizeof(float));

    // Copy host input vector to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    // Execute the kernel
    cuda_vector_mbp_reduce<<<gridSize, blockSize>>>(d_a, d_out, n);
    //dev_out now holds the partial result
    cuda_vector_mbp_reduce<<<1, blockSize>>>(d_out, d_out, blockSize);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();

    // Copy array back to host --- just the first element
    hipMemcpy(b, d_out, sizeof(float), hipMemcpyDeviceToHost );

    // Release device memory
    hipFree(d_a);
    hipFree(d_out);
};
